
#include <hip/hip_runtime.h>
#include <stdio.h>

void helloCPU()
{
  printf("Hello from the CPU.\n");
}

__global__ void helloGPU()
{
  printf("Hello from the GPU.\n");
}

int main(void)
{

  helloCPU();

  helloGPU<<<1, 1>>>();
   
  hipDeviceSynchronize();
   
  return 0;
}
