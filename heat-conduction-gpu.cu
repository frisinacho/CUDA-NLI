#include "hip/hip_runtime.h"
#define I2D(num, c, r) ((r)*(num)+(c))
__global__ void kernel(int ni, int nj, float fact, float* temp_in, float*temp_out)
{
	// loop over all points in domain (except boundary) (0,0)
	j = blockIdx.x * blockDim.x + threadIdx.x;
	i = blockIdx.y * blockDim.y + threadIdx.y;
	if (((j>0) && (i>0)) && ((j<nj-1) && (i<ni-1))) {
		// find indices into linear memory
		// for central point and neighbours
		int i00 = I2D(ni, i, j);
		int im10 = I2D(ni, i-1, j);
		int ip10 = I2D(ni, i+1, j);
		int i0m1 = I2D(ni, i, j-1);
		int i0p1 = I2D(ni, i, j+1);

		// evaluate derivatives
		float d2tdx2 = temp_in[im10]-2*temp_in[i00]+temp_in[ip10];
		float d2tdy2 = temp_in[i0m1]-2*temp_in[i00]+temp_in[i0p1];
		
		// update temperatures
		temp_out[i00] = temp_in[i00]+fact*(d2tdx2 + d2tdy2);
	}
}
